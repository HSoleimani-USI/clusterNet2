#include <basicOps.cuh>
#include <clusterKernels.cuh>
#include <hdf5.h>

template Matrix<int> *Matrix<int>::to_host();
template Matrix<float> *Matrix<float>::to_host();
template <typename T> Matrix<T> *Matrix<T>::to_host()
{
	Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
	T *cpu_data;

	cpu_data = (T*)malloc(bytes);
	CUDA_CHECK_RETURN(hipMemcpy(cpu_data,data,bytes,hipMemcpyDefault));
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->data = cpu_data;
  
  return out;
}


template void Matrix<float>::free_matrix();
template <typename T> void Matrix<T>::free_matrix()
{
	CUDA_CHECK_RETURN(hipFree(data));
	free(this);
}

//to host where we already have created a gpu buffer
template void to_host(Matrix<int> *gpu, int *cpu);
template void to_host(Matrix<float> *gpu, float *cpu);
template <typename T> void to_host(Matrix<T> *gpu, T *cpu)
{ CUDA_CHECK_RETURN(hipMemcpy(cpu,gpu->data,gpu->bytes,hipMemcpyDefault)); }

//pinned memory needed for asynchronous copies between CPU and GPU
//pinned memory makes sure that we do not have to allocate a page CPU buffer before the copy
//this makes the copy faster and asynchronous with respect to the caller
template Matrix<int> *to_pinned(int rows, int cols, int *cpu);
template Matrix<float> *to_pinned(int rows, int cols, float *cpu);
template <typename T> Matrix<T> *to_pinned(int rows, int cols, T *cpu){ return to_pinned<T>(rows, cols, cpu,sizeof(T)*rows*cols); }

template Matrix<int> *to_pinned(int rows, int cols, int *cpu, size_t bytes_to_copy);
template Matrix<float> *to_pinned(int rows, int cols, float *cpu, size_t bytes_to_copy);
template <typename T> Matrix<T> *to_pinned(int rows, int cols, T *cpu, size_t bytes_to_copy)
{
	int size = rows*cols;
	size_t bytes = sizeof(T)*size;
	Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
	T *pinned_ptr;
	CUDA_CHECK_RETURN(hipHostAlloc(&pinned_ptr, bytes, hipHostMallocPortable));
	for(int i = 0; i < rows*cols; i++){ pinned_ptr[i] = 0.0f;}
	CUDA_CHECK_RETURN(hipMemcpy(pinned_ptr,cpu,bytes_to_copy,hipMemcpyDefault));

	out->bytes = bytes;
	out->size = size;
	out->rows = rows;
	out->cols = cols;
	out->data = pinned_ptr;

	return out;
}

template Matrix<float> *zeros<float>(int rows, int cols);
template <typename T> Matrix<T> *zeros(int rows, int cols)
{
	return fill_matrix<T>(rows, cols, (T)0.0f);
}


template Matrix<float> *ones<float>(int rows, int cols);
template <typename T> Matrix<T> *ones(int rows, int cols)
{
	return fill_matrix<T>(rows, cols, (T)1.0f);
}

template Matrix<int> *empty<int>(int rows, int cols);
template Matrix<float> *empty<float>(int rows, int cols);
template <typename T> Matrix<T> *empty(int rows, int cols)
{
  T *gpu_data;
  int size = rows*cols;
  size_t bytes = rows*cols*sizeof(T);
  CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, bytes));
  
  Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
  out->rows = rows;
  out->cols = cols;
  out->bytes = bytes;
  out->size = size;
  out->data = gpu_data;

  return out;
}

template void to_gpu(unsigned int *cpu, Matrix<unsigned int> *gpu);
template void to_gpu(int *cpu, Matrix<int> *gpu);
template void to_gpu(float *cpu, Matrix<float> *gpu);
template<typename T> void to_gpu(T *cpu, Matrix<T> *gpu)
{
    CUDA_CHECK_RETURN(hipMemcpy(gpu->data,cpu,gpu->bytes,hipMemcpyDefault));
}


template Matrix<unsigned long long> *fill_matrix(int rows, int cols, unsigned long long fill_value);
template Matrix<unsigned int> *fill_matrix(int rows, int cols, unsigned int fill_value);
template Matrix<int> *fill_matrix(int rows, int cols, int fill_value);
template Matrix<float> *fill_matrix(int rows, int cols, float fill_value);
template <typename T> Matrix<T> *fill_matrix(int rows, int cols, T fill_value)
{
  if(rows < 1 || cols < 1)
  {
    printf("Error: Dimensions must be greater than zero!\n");
  }
 
  Matrix<T> *out = empty<T>(rows, cols);

  kFill_with<T><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(out->data,fill_value,out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
 
  return out;
}

template void sortbykey(Matrix<float> *keys, Matrix<float> *values);
template <typename T> void sortbykey(Matrix<T> *keys, Matrix<T> *values)
{
	thrust::device_ptr<T> d_values(values->data);
	thrust::device_ptr<T> d_keys(keys->data);

	thrust::sort_by_key(d_keys, d_keys + keys->size, d_values);
}


float sum(Matrix<float> *A)
{
	thrust::device_ptr<float> d_values(A->data);
	return thrust::reduce(d_values, d_values+A->size);
}


template void transpose(Matrix<float> *A, Matrix<float> *out, int rows, int cols);
template <typename T> void transpose(Matrix<T> *A, Matrix<T> *out, int rows, int cols)
{
  // setup execution parameters
  int grid_x = rows / COPY_BLOCK_SIZE;
  if (rows % COPY_BLOCK_SIZE)
    grid_x++;

  int grid_y = cols / COPY_BLOCK_SIZE;
  if (cols % COPY_BLOCK_SIZE)
    grid_y++;

  dim3 grid(grid_x, grid_y, 1);
  dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
  kTranspose<T><<< grid, threads >>>(A->data, out->data, rows, cols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());

}

//the column major format has increasing indexes along its columns:
/*
 * 			[0 3 6]
 * 			[1 4 7]
 * 			[2 5 8]
 */

template Matrix<float> *to_col_major(Matrix<float> *A);
template <typename T> Matrix<T> *to_col_major(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->rows,A->cols);
  transpose<T>(A, out, A->cols,A->rows);
  return out;
}

template void to_col_major(Matrix<unsigned int> *A, Matrix<unsigned int> *out);
template void to_col_major(Matrix<float> *A, Matrix<float> *out);
template <typename T> void to_col_major(Matrix<T> *A, Matrix<T> *out)
{
	transpose<T>(A, out, A->cols,A->rows);
}

//the row major format has increasing indexes along its rows:
/*
 * 			[0 1 2]
 * 			[3 4 5]
 * 			[6 7 8]
 */
template Matrix<float> *to_row_major(Matrix<float> *A);
template <typename T> Matrix<T> *to_row_major(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->rows,A->cols);
  transpose<T>(A, out, A->rows,A->cols);

  return out;
}

template Matrix<unsigned int> *transpose(Matrix<unsigned int> *A);
template Matrix<float> *transpose(Matrix<float> *A);
template <typename T> Matrix<T> *transpose(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->cols,A->rows);
  transpose<T>(A, out, A->rows,A->cols);

  out->rows = A->cols;
  out->cols = A->rows;
  return out;
}

//elementwise operation with a single matrix argument
template void elementWise<kabs>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<klog>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<ksqrt>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<klogistic>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<klogistic_grad>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<kELU>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<kELU_grad>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<krectified>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<krectified_grad>(Matrix<float> *A, Matrix<float>*out);
template void elementWise<kcopy>(Matrix<float> *A, Matrix<float>*out);
template <int action> void elementWise(Matrix<float> *A, Matrix<float>*out)
{
  check_for_same_dimensions(A,out);
  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, NULL, out->data,0.0f, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void elementWise<kpow>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWise<ksmul>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWise<kssub>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWise<ksgt>(Matrix<float> *A, Matrix<float>*out, float scalar);
template <int action> void elementWise(Matrix<float> *A, Matrix<float>*out, float scalar)
{
  check_for_same_dimensions(A,out);
  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, NULL, out->data,scalar, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//elementwise operation with a two matrix arguments
template void elementWise<kadd>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<ksub>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kdiv>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kmul>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<keq>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<klt>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kgt>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kge>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kle>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<kne>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template void elementWise<ksquared_diff>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out);
template <int action> void elementWise(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out)
{
  check_for_same_dimensions(A,B);
  check_for_same_dimensions(A,out);

  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, B->data, out->data,0.0f, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void elementWise<kdropout>(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out, float scalar);
template <int action> void elementWise(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out, float scalar)
{
  check_for_same_dimensions(A,B);
  check_for_same_dimensions(A,out);


  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, B->data, out->data,scalar, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//vectorwise operation between matrix and vector
//this is equivalent to broadcasting in numpy
template void vectorWise<kvadd>(Matrix<float> *A, Matrix<float> *v, Matrix<float>*out);
template void vectorWise<kvsub>(Matrix<float> *A, Matrix<float> *v, Matrix<float>*out);
template <int action> void vectorWise(Matrix<float> *A, Matrix<float> *v, Matrix<float>*out)
{
  check_matrix_vector_op(A, v);
  kVectorWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, v->data, out->data, 0.0f, out->rows, out->cols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void vectorWise<ktmatrix>(Matrix<float> *v, Matrix<float>*out);
template <int action> void vectorWise(Matrix<float> *v, Matrix<float>*out)
{
  check_matrix_vector_op(out, v);
  kVectorWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(NULL, v->data, out->data, 0.0f, out->rows, out->cols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//slice rows and columns
//equivalent to python slicing, e.h. X[3:4,6:9] is equivalent to slice(X,out, 3,4,6,9)
void slice(Matrix<float> *A, Matrix<float>*out, int rstart, int rend, int cstart, int cend)
{
  kSlice<<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, out->data, A->rows, A->cols, rstart, rend, cstart, cend);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void reduceToRows<rmax>(Matrix<float> *A, Matrix<float> *vout);
template void reduceToRows<rsum>(Matrix<float> *A, Matrix<float> *vout);
template void reduceToRows<rmean>(Matrix<float> *A, Matrix<float> *vout);
template <int reduction> void reduceToRows(Matrix<float> *A, Matrix<float> *vout)
{
    kReduceToRows<reduction><<<A->rows > 1024 ? 1024 : A->rows, 256>>>(A->data, vout->data, A->rows, A->cols);
    CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void reduceToCols<rmax>(Matrix<float> *A, Matrix<float> *vout);
template void reduceToCols<rsum>(Matrix<float> *A, Matrix<float> *vout);
template void reduceToCols<rmean>(Matrix<float> *A, Matrix<float> *vout);
template <int reduction> void reduceToCols(Matrix<float> *A, Matrix<float> *vout)
{
	kReduceToCols<reduction><<<A->cols > 1024 ? 1024 : A->cols, 32>>>(A->data, vout->data, A->rows, A->cols);
    CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template float reduceToValue<rsum>(Matrix<float> *A);
template float reduceToValue<rmax>(Matrix<float> *A);
template float reduceToValue<rmean>(Matrix<float> *A);
template <int reduction> float reduceToValue(Matrix<float> *A)
{
	Matrix<float> *vout = empty<float>(A->rows, 1);
	float retValue = reduceToValue<reduction>(A, vout);
	CUDA_CHECK_RETURN(hipFree(vout->data));
	free(vout);
	return retValue;
}

template float reduceToValue<rsum>(Matrix<float> *A, Matrix<float> *vout_rows);
template float reduceToValue<rmax>(Matrix<float> *A, Matrix<float> *vout_rows);
template float reduceToValue<rmean>(Matrix<float> *A, Matrix<float> *vout_rows);
template <int reduction> float reduceToValue(Matrix<float> *A, Matrix<float> *vout_rows)
{
	reduceToRows<reduction>(A, vout_rows);
	Matrix<float> *value = empty<float>(1,1);
    kReduceToRows<reduction><<<1, 256>>>(vout_rows->data, value->data, 1, A->rows);
    CUDA_CHECK_RETURN(hipPeekAtLastError());

    float retValue = 0.0f;

	CUDA_CHECK_RETURN(hipMemcpy(&retValue,value->data,value->bytes,hipMemcpyDefault));

	hipFree(value->data);
	free(value);

    return retValue;
}

//this softmax is numerically stable
void softmax(Matrix<float> *A, Matrix<float> *out)
{
	check_for_same_dimensions(A, out);
    kSoftMax<<<A->rows > 1024 ? 1024 : A->rows, 256>>>(A->data, out->data, A->rows, A->cols);
    CUDA_CHECK_RETURN(hipPeekAtLastError());
}


void argmax(Matrix<float> *A, Matrix<float> *out)
{
	check_matrix_vector_op(A, out);
    kArgmax<<<A->rows > 1024 ? 1024 : A->rows, 256>>>(A->data, out->data, A->rows, A->cols);
    CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void WeightUpdate<RMSProp>(Matrix<float> *RMS, Matrix<float> *grad, Matrix<float> *w, float RMS_multiplier, float learning_rate);
template void WeightUpdate<RMSPropInit>(Matrix<float> *RMS, Matrix<float> *grad, Matrix<float> *w, float RMS_multiplier, float learning_rate);
template <int action> void WeightUpdate(Matrix<float> *RMS, Matrix<float> *grad, Matrix<float> *w, float RMS_multiplier, float learning_rate)
{
	check_for_same_dimensions(RMS, grad);
	check_for_same_dimensions(RMS, w);
	int threads = 256;
	int blocks = (RMS->size/threads) + 1;
	kRMSprop<action><<<blocks,threads>>>(RMS->data, grad->data, w->data, RMS_multiplier, learning_rate, RMS->size);
    CUDA_CHECK_RETURN(hipPeekAtLastError());
}

Matrix<float> *read_hdf5(const char *filepath){ return read_hdf5(filepath,"/Default"); }
Matrix<float> *read_hdf5(const char *filepath, const char *tag)
{
	   hid_t       file_id, dataset_id;

	   file_id = H5Fopen(filepath, H5F_ACC_RDWR, H5P_DEFAULT);
	   dataset_id = H5Dopen2(file_id, tag, H5P_DEFAULT);

	   hid_t dspace = H5Dget_space(dataset_id);
	   hsize_t dims[2];
	   H5Sget_simple_extent_dims(dspace, dims, NULL);
	   size_t bytes = sizeof(float)*dims[0]*dims[1];

	   float *data;
	   CUDA_CHECK_RETURN(hipHostAlloc(&data, bytes, hipHostMallocPortable));

	   H5Dread(dataset_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, data);
	   H5Dclose(dataset_id);
	   H5Fclose(file_id);

	   Matrix<float> *out = (Matrix<float>*)malloc(sizeof(Matrix<float>));
	   out->rows = (int)dims[0];
	   out->cols= (int)dims[1];
	   out->bytes = bytes;
	   out->data = data;
	   out->size = (int)(dims[0]*dims[1]);

	   return out;
}

bool check_for_same_dimensions(Matrix<float> *A, Matrix<float> *B)
{
	if(A && B)
	{
		if(A->rows == B->rows && A->cols == B->cols) return true;
		else
		{
			cout << "Matrices do not have the same dimension: " << A->rows << "x" << A->cols << " vs " << B->rows << "x" << B->cols << endl;
			throw "Matricies do not have same dimension!";
		}
	}
	else
		return true;
}

bool check_matrix_multiplication(Matrix<float> *A, Matrix<float> *B, Matrix<float> *out, bool T1, bool T2)
{
	int A_rows = A->rows, A_cols = A->cols, B_rows = B->rows, B_cols = B->cols;
	if (T1){ A_rows = A->cols; A_cols = A->rows; }
	if (T2){ B_rows = B->cols; B_cols = B->rows; }

	if(A_rows == out->rows && A_cols == B_rows && B_cols == out->cols) return true;
	else
	{
		cout << "Matrices are not aligned: " << A_rows<< "x" << A_cols << " dot " << B_rows << "x" << B_cols << " -->"  << out->rows << "x" << out->cols <<endl;
		throw "Matrices are not aligned!";
	}

}

bool check_matrix_vector_op(Matrix<float> *A, Matrix<float> *vec)
{
	if(A && vec)
	{
		if((A->rows == vec->rows && vec->cols == 1) ||
		   (A->cols == vec->rows && vec->cols == 1) ||
		   (A->rows == vec->cols && vec->rows == 1) ||
		   (A->cols == vec->cols && vec->rows == 1)) return true;
		else
		{
			cout << "Matrix vector opt does not align: " << A->rows << "x" << A->cols << " vs " << vec->rows << "x" << vec->cols << endl;
			throw "Matrix vector opt does not align!";
		}
	}
	else return true;
}




void print_matrix(Matrix<float> *A, int end_rows, int end_cols){ print_matrix(A,0,end_rows,0,end_cols); }
void print_matrix(Matrix<float> *A, int start_row, int end_row, int start_col, int end_col)
{
	for(int row = start_row; row< end_row; row++)
	{
		printf("[");
		for(int col =start_col; col < end_col; col++)
		{
		  if(A->data[(row*A->cols)+col] < 0.0f)
			  printf("% f ",A->data[(row*A->cols)+col]);
		  else
			  printf("%f ",A->data[(row*A->cols)+col]);
		}
		printf("]\n");
	}
	printf("\n");
}

void printmat(Matrix<float> *A)
{
  Matrix<float> * m = A->to_host();
  print_matrix(m,A->rows,A->cols);
  free(m->data);
  free(m);

}

void printdim(Matrix<float> *A)
{
	cout << A->rows << "x" << A->cols << endl;
}

void printsum(Matrix<float> *A)
{
	cout << sum(A) << endl;
}

void printhostmat(Matrix<float> *A){ print_matrix(A,A->rows,A->cols); }
void printmat(Matrix<float> *A, int end_rows, int end_cols)
{
  Matrix<float> * m = A->to_host();
  print_matrix(m, end_rows, end_cols);
  free(m->data);
  free(m);

}

void printmat(Matrix<float> *A, int start_row, int end_row, int start_col, int end_col)
{
  Matrix<float> * m = A->to_host();
  print_matrix(m, start_row, end_row, start_col, end_col);
  free(m->data);
  free(m);

}

Matrix<float> *get_view(Matrix<float> *A, int rstart, int rend)
{
	assert(rstart < A->rows);
	assert(rstart >= 0);
	assert(rend <= A->rows);

	Matrix<float> *ret = new Matrix<float>();
	ret->rows = rend-rstart;
	ret->cols = A->cols;
	ret->size = ret->rows*ret->cols;
	ret->bytes = sizeof(float)*ret->size;

	ret->data = &(A->data)[rstart*A->cols];

	return ret;
}
