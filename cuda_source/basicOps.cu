#include <basicOps.cuh>
#include <clusterKernels.cuh>
#include <iostream>     // std::cout

using std::cout;
using std::endl;

template Matrix<int> *Matrix<int>::to_host();
template Matrix<float> *Matrix<float>::to_host();
template <typename T> Matrix<T> *Matrix<T>::to_host()
{
	Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
	T *cpu_data;

	cpu_data = (T*)malloc(bytes);
	CUDA_CHECK_RETURN(hipMemcpy(cpu_data,data,bytes,hipMemcpyDefault));
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->data = cpu_data;
  
  return out;
}


template void to_host(Matrix<int> *gpu, int *cpu);
template void to_host(Matrix<float> *gpu, float *cpu);
template <typename T> void to_host(Matrix<T> *gpu, T *cpu)
{ CUDA_CHECK_RETURN(hipMemcpy(cpu,gpu->data,gpu->bytes,hipMemcpyDefault)); }

template Matrix<int> *to_pinned(int rows, int cols, int *cpu);
template Matrix<float> *to_pinned(int rows, int cols, float *cpu);
template <typename T> Matrix<T> *to_pinned(int rows, int cols, T *cpu)
{
	int size = rows*cols;
	size_t bytes = sizeof(T)*size;
	Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
	T *pinned_ptr;
	hipHostAlloc(&pinned_ptr, bytes, hipHostMallocPortable);
	CUDA_CHECK_RETURN(hipMemcpy(pinned_ptr,cpu,bytes,hipMemcpyDefault));

	out->bytes = bytes;
	out->size = size;
	out->rows = rows;
	out->cols = cols;
	out->data = pinned_ptr;

	return out;
}


template Matrix<int> *empty<int>(int rows, int cols);
template Matrix<float> *empty<float>(int rows, int cols);
template <typename T> Matrix<T> *empty(int rows, int cols)
{
  T *gpu_data;
  int size = rows*cols;
  size_t bytes = rows*cols*sizeof(T);
  CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, bytes));
  
  Matrix<T> *out = (Matrix<T>*)malloc(sizeof(Matrix<T>));
  out->rows = rows;
  out->cols = cols;
  out->bytes = bytes;
  out->size = size;
  out->data = gpu_data;

  return out;
}

template void to_gpu(unsigned int *cpu, Matrix<unsigned int> *gpu);
template void to_gpu(int *cpu, Matrix<int> *gpu);
template void to_gpu(float *cpu, Matrix<float> *gpu);
template<typename T> void to_gpu(T *cpu, Matrix<T> *gpu)
{
    CUDA_CHECK_RETURN(hipMemcpy(gpu->data,cpu,gpu->bytes,hipMemcpyDefault));
  	//to_col_major<T>(gpu,gpu);
}


template Matrix<unsigned long long> *fill_matrix(int rows, int cols, unsigned long long fill_value);
template Matrix<unsigned int> *fill_matrix(int rows, int cols, unsigned int fill_value);
template Matrix<int> *fill_matrix(int rows, int cols, int fill_value);
template Matrix<float> *fill_matrix(int rows, int cols, float fill_value);
template <typename T> Matrix<T> *fill_matrix(int rows, int cols, T fill_value)
{
  if(rows < 1 || cols < 1)
  {
    printf("Error: Dimensions must be greater than zero!\n");
  }
 
  Matrix<T> *out = empty<T>(rows, cols);

  kFill_with<T><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(out->data,fill_value,out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
 
  return out;
}


template void transpose(Matrix<float> *A, Matrix<float> *out, int rows, int cols);
template <typename T> void transpose(Matrix<T> *A, Matrix<T> *out, int rows, int cols)
{
  // setup execution parameters
  int grid_x = rows / COPY_BLOCK_SIZE;
  if (rows % COPY_BLOCK_SIZE)
    grid_x++;

  int grid_y = cols / COPY_BLOCK_SIZE;
  if (cols % COPY_BLOCK_SIZE)
    grid_y++;

  dim3 grid(grid_x, grid_y, 1);
  dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
  kTranspose<T><<< grid, threads >>>(A->data, out->data, rows, cols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());

}

template Matrix<float> *to_col_major(Matrix<float> *A);
template <typename T> Matrix<T> *to_col_major(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->rows,A->cols);
  transpose<T>(A, out, A->cols,A->rows);
  return out;
}

template void to_col_major(Matrix<unsigned int> *A, Matrix<unsigned int> *out);
template void to_col_major(Matrix<float> *A, Matrix<float> *out);
template <typename T> void to_col_major(Matrix<T> *A, Matrix<T> *out)
{
	transpose<T>(A, out, A->cols,A->rows);
}

template Matrix<float> *to_row_major(Matrix<float> *A);
template <typename T> Matrix<T> *to_row_major(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->rows,A->cols);
  transpose<T>(A, out, A->rows,A->cols);

  return out;
}

template Matrix<unsigned int> *transpose(Matrix<unsigned int> *A);
template Matrix<float> *transpose(Matrix<float> *A);
template <typename T> Matrix<T> *transpose(Matrix<T> *A)
{
  Matrix<T> *out = empty<T>(A->cols,A->rows);
  transpose<T>(A, out, A->rows,A->cols);

  out->rows = A->cols;
  out->cols = A->rows;
  return out;
}



template void elementWiseUnary<kabs>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<klog>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<ksqrt>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<kpow>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<klogistic>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<klogistic_grad>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<krectified>(Matrix<float> *A, Matrix<float>*out, float scalar);
template void elementWiseUnary<krectified_grad>(Matrix<float> *A, Matrix<float>*out, float scalar);
template <int action> void elementWiseUnary(Matrix<float> *A, Matrix<float>*out, float scalar)
{
  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, NULL, out->data,scalar, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void elementWise<kadd>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<ksub>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kdiv>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kmul>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<keq>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<klt>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kgt>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kge>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kle>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<kne>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template void elementWise<ksquared_diff>(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar);
template <int action> void elementWise(Matrix<float> *A, Matrix<float> *B, Matrix<float>*out, float scalar)
{
  kElementWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, B->data, out->data,scalar, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template void vectorWise<kvadd>(Matrix<float> *A, Matrix<float> *v, Matrix<float>*out, float scalar);
template <int action> void vectorWise(Matrix<float> *A, Matrix<float> *v, Matrix<float>*out, float scalar)
{
  kVectorWise<action><<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, v->data, out->data, scalar, out->cols, out->size);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void slice(Matrix<float> *A, Matrix<float>*out, int rstart, int rend, int cstart, int cend)
{
  kSlice<<<out->size/THREADS_PER_BLOCKS + 1, THREADS_PER_BLOCKS>>>(A->data, out->data, A->rows, A->cols, rstart, rend, cstart, cend);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void softmax(Matrix<float> *A, Matrix<float> *out)
{
    kSoftMax<<<A->rows > 1024 ? 1024 : A->rows, 256>>>(A->data, out->data, A->rows, A->cols);
    CUDA_CHECK_RETURN(hipPeekAtLastError());

}
