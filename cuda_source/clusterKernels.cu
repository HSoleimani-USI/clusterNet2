#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
#include <math.h>
#include <basicOps.cuh>

template __global__ void kTranspose<float>(const float *A, float *out, int width, int height);
template __global__ void kTranspose<int>(const int *A, int *out, int width, int height);
template __global__ void kTranspose<unsigned int>(const unsigned int *A, unsigned int *out, int width, int height);
template<typename T> __global__ void kTranspose(const T *A, T *out, int width, int height)
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the Matrix *tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed Matrix *tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

template __global__ void kFill_with<int>(int *m, const int fill_value, int size);
template __global__ void kFill_with<float>(float *m, const float fill_value, int size);
template __global__ void kFill_with<unsigned int>(unsigned int *m, const unsigned int fill_value, int size);
template __global__ void kFill_with<unsigned long long>(unsigned long long *m, const unsigned long long fill_value, int size);
template<typename T> __global__ void kFill_with(T *m, const T fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}



template __global__ void kElementWise<kabs>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<klog>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<ksqrt>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<kpow>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kadd>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<ksub>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kdiv>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kmul>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klogistic>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klogistic_grad>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<krectified>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<krectified_grad>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<keq>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klt>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kgt>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kle>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kge>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kne>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<ksquared_diff>(const float *A, const float *B, float *out, const float scalar, int size);
template<int operation> __global__ void kElementWise(const float *A, const float *B, float *out, const float scalar, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       switch(operation)
	   {
       	   case kabs: out[i] = fabsf(A[i]); break;
       	   case klog: out[i] = __logf(A[i]); break;
       	   case ksqrt: out[i] = sqrtf(A[i]); break;
       	   case kpow: out[i] = powf(A[i],scalar); break;
       	   case kadd: out[i] = A[i] + B[i]; break;
       	   case ksub: out[i] = A[i] - B[i]; break;
       	   case kdiv: out[i] = fdividef(A[i], B[i]); break;
       	   case kmul: out[i] = A[i] * B[i]; break;
       	   case klogistic: out[i] = 1.0f/(1.0f + expf(A[i])); break;
       	   case klogistic_grad: out[i] = A[i]*(A[i]-1.0f); break;
       	   case krectified: out[i] = A[i] > 0.0f ? A[i] : 0.0f; break;
       	   case krectified_grad: out[i] = A[i] > 0.0f ? 1.0f : 0.0f; break;
       	   case keq: out[i] = (float)(A[i] == B[i]); break;
       	   case klt: out[i] = (float)(A[i] < B[i]); break;
       	   case kgt: out[i] = (float)(A[i] > B[i]); break;
       	   case kge: out[i] = (float)(A[i] >= B[i]); break;
       	   case kle: out[i] = (float)(A[i] <= B[i]); break;
       	   case kne: out[i] = (float)(A[i] != B[i]); break;
       	   case ksquared_diff: out[i] = powf(A[i]-B[i],2.0f); break;
	   }
  }
}

template __global__ void kVectorWise<kvadd>(float *A, float *v, float *out, const float scalar, int rows, int size);
template <int operation> __global__ void kVectorWise(float *A, float *v, float *out, const float scalar, int rows, int size)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	int offset = 0;
	for (unsigned int i = idx;i < size; i += numThreads)
	{
		offset = (i / rows);
		switch(operation)
		{
			case kvadd: out[i] =  A[i] + v[offset]; break;
		}
	}
}


//for column major data
__global__ void kSlice(float *A, float *out, int rows_A, int cols_A, int rstart, int rend, int cstart, int cend)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int rows_out = (rend - rstart);
  int cols_out = (cend - cstart);
  int size = rows_out*cols_out;

  int current_col = 0;
  int offset = 0;
  int current_row = 0;
  for (unsigned int i = idx;i < size; i += numThreads)
  {
	  current_col = i / rows_out;
	  current_row = i - (current_col*rows_out);

	  offset = (rows_A*(current_col+cstart)) + current_row + rstart;
	  out[i] = A[offset];
  }
}

__global__ void kSoftMax(float* A, float* out, const unsigned int rows, const unsigned int cols)
{
	float col_value = 0.0f;
	const unsigned int idx = threadIdx.x*threadIdx.y;

	__shared__ float max_values[32][32];
	__shared__ float row_sums[32][32];

	for (unsigned int row = blockIdx.x; row < rows; row += gridDim.x)
	{
		//fill with min values
		max_values[threadIdx.x][threadIdx.y] = -FLT_MAX;
		row_sums[threadIdx.x][threadIdx.y] = 0.0f;

		 //calc max value of the row
		for (unsigned int i = idx; i < cols; i+=blockDim.x)
		{
			col_value = A[(i*rows) + row];
			max_values[threadIdx.x][threadIdx.y] = fmaxf(max_values[threadIdx.x][threadIdx.y],col_value);
			row_sums[threadIdx.x][threadIdx.y] += col_value;
		}

		reduce<0>(row_sums[threadIdx.x],idx,blockDim.x);
		reduce<1>(max_values[threadIdx.x],idx,blockDim.x);


		//calc the value of each element in the row
		for (unsigned int i = idx; i < cols; i+=blockDim.x)
		{
			out[(i*rows) + row] = __expf(A[(i*rows) + row] - max_values[0][0])/row_sums[0][0];
		}

	}
}

template __device__ float reduction_action<0>(float a, float b);
template __device__ float reduction_action<1>(float a, float b);
template<int action> __device__ float reduction_action(float a, float b)
{
	switch(action)
	{
		case 0: return a+b;
		case 1: return fmaxf(a,b);
	}
}

template __device__ void reduce<0>(float* sdata, const unsigned int tid, const unsigned int threads);
template __device__ void reduce<1>(float* sdata, const unsigned int tid, const unsigned int threads);
template <int action> __device__ void reduce(float* sdata, const unsigned int tid, const unsigned int threads)
{

	  //Synchronize threads to share shared memory data
	  __syncthreads();

	  float agg = sdata[tid];

	  // do reduction in shared mem
	  if (threads >= 1024) { if (tid < 512) { sdata[tid] = agg = reduction_action<action>(agg, sdata[tid + 512]); } __syncthreads(); }
	  if (threads >= 512) { if (tid < 256) { sdata[tid] = agg = reduction_action<action>(agg, sdata[tid + 256]); } __syncthreads(); }
	  if (threads >= 256) { if (tid < 128) { sdata[tid] = agg = reduction_action<action>(agg, sdata[tid + 128]); } __syncthreads(); }
	  if (threads >= 128) { if (tid <  64) { sdata[tid] = agg = reduction_action<action>(agg, sdata[tid + 64]);  } __syncthreads(); }

	  if (threads == 32){
	    if (tid < 16)
	    {
	      // now that we are using warp-synchronous programming (below)
	      // we need to declare our shared memory volatile so that the compiler
	      // doesn't reorder stores to it and induce incorrect behavior.
	      volatile float* smem = sdata;
	      if (threads >=  32) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 16]); }
	      if (threads >=  16) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 8]); }
	      if (threads >=   8) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 4]);; }
	      if (threads >=   4) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 2]); }
	      if (threads >=   2) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 1]); }
	    }
	  }
	  else
	  {
	    if (tid < 32)
	    {
	      // now that we are using warp-synchronous programming (below)
	      // we need to declare our shared memory volatile so that the compiler
	      // doesn't reorder stores to it and induce incorrect behavior.
	      volatile float* smem = sdata;
	      if (threads >=  64) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 32]); }
	      if (threads >=  32) { smem[tid] = agg = reduction_action<action>(agg, smem[tid + 16]); }
	      if (threads >=  16) { smem[tid] = agg = agg = reduction_action<action>(agg, smem[tid + 8]); }
	      if (threads >=   8) { smem[tid] = agg = agg = reduction_action<action>(agg, smem[tid + 4]); }
	      if (threads >=   4) { smem[tid] = agg = agg = reduction_action<action>(agg, smem[tid + 2]); }
	      if (threads >=   2) { smem[tid] = agg = agg = reduction_action<action>(agg, smem[tid + 1]); }
	    }
	  }

	  __syncthreads();

}


