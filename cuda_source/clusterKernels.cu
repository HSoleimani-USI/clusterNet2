#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
#include <math.h>

template __global__ void kTranspose<float>(const float *A, float *out, int width, int height);
template __global__ void kTranspose<int>(const int *A, int *out, int width, int height);
template __global__ void kTranspose<unsigned int>(const unsigned int *A, unsigned int *out, int width, int height);
template<typename T> __global__ void kTranspose(const T *A, T *out, int width, int height)
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the Matrix *tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed Matrix *tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

template __global__ void kFill_with<int>(int *m, const int fill_value, int size);
template __global__ void kFill_with<float>(float *m, const float fill_value, int size);
template __global__ void kFill_with<unsigned int>(unsigned int *m, const unsigned int fill_value, int size);
template __global__ void kFill_with<unsigned long long>(unsigned long long *m, const unsigned long long fill_value, int size);
template<typename T> __global__ void kFill_with(T *m, const T fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}



template __global__ void kElementWise<0>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<1>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<2>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<3>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<4>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<5>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<6>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<7>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<8>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<9>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<10>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<11>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<12>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<13>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<14>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<15>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<16>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<17>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<18>(const float *A, const float *B, float *out, const float scalar, int size);
template<int operation> __global__ void kElementWise(const float *A, const float *B, float *out, const float scalar, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       switch(operation)
	   {
       	   case 0: out[i] = fabsf(A[i]); break;
       	   case 1: out[i] = __logf(A[i]); break;
       	   case 2: out[i] = sqrtf(A[i]); break;
       	   case 3: out[i] = powf(A[i],scalar); break;
       	   case 4: out[i] = A[i] + B[i]; break;
       	   case 5: out[i] = A[i] - B[i]; break;
       	   case 6: out[i] = fdividef(A[i], B[i]); break;
       	   case 7: out[i] = A[i] * B[i]; break;
       	   case 8: out[i] = 1.0f/(1.0f + expf(A[i])); break;
       	   case 9: out[i] = A[i]*(A[i]-1.0f); break;
       	   case 10: out[i] = A[i] > 0.0f ? A[i] : 0.0f; break;
       	   case 11: out[i] = A[i] > 0.0f ? 1.0f : 0.0f; break;
       	   case 12: out[i] = (float)(A[i] == B[i]); break;
       	   case 13: out[i] = (float)(A[i] < B[i]); break;
       	   case 14: out[i] = (float)(A[i] > B[i]); break;
       	   case 15: out[i] = (float)(A[i] >= B[i]); break;
       	   case 16: out[i] = (float)(A[i] <= B[i]); break;
       	   case 17: out[i] = (float)(A[i] != B[i]); break;
       	   case 18: out[i] = powf(A[i]-B[i],2.0f); break;
	   }
  }
}



