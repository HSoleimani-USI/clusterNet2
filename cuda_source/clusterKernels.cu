#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>

template __global__ void kTranspose<float>(const float *A, float *out, int width, int height);
template __global__ void kTranspose<int>(const int *A, int *out, int width, int height);
template __global__ void kTranspose<unsigned int>(const unsigned int *A, unsigned int *out, int width, int height);
template<typename T> __global__ void kTranspose(const T *A, T *out, int width, int height)
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the Matrix *tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed Matrix *tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

template __global__ void kFill_with<int>(int *m, const int fill_value, int size);
template __global__ void kFill_with<float>(float *m, const float fill_value, int size);
template __global__ void kFill_with<unsigned int>(unsigned int *m, const unsigned int fill_value, int size);
template __global__ void kFill_with<unsigned long long>(unsigned long long *m, const unsigned long long fill_value, int size);
template<typename T> __global__ void kFill_with(T *m, const T fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}




