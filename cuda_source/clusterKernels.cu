#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
#include <math.h>
#include <basicOps.cuh>

template __global__ void kTranspose<float>(const float *A, float *out, int width, int height);
template __global__ void kTranspose<int>(const int *A, int *out, int width, int height);
template __global__ void kTranspose<unsigned int>(const unsigned int *A, unsigned int *out, int width, int height);
template<typename T> __global__ void kTranspose(const T *A, T *out, int width, int height)
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the Matrix *tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed Matrix *tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

template __global__ void kFill_with<int>(int *m, const int fill_value, int size);
template __global__ void kFill_with<float>(float *m, const float fill_value, int size);
template __global__ void kFill_with<unsigned int>(unsigned int *m, const unsigned int fill_value, int size);
template __global__ void kFill_with<unsigned long long>(unsigned long long *m, const unsigned long long fill_value, int size);
template<typename T> __global__ void kFill_with(T *m, const T fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}



template __global__ void kElementWise<kabs>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<klog>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<ksqrt>(const float *A, const float *B,  float *out, const float scalar, int size);
template __global__ void kElementWise<kpow>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kadd>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<ksub>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kdiv>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kmul>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klogistic>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klogistic_grad>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<krectified>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<krectified_grad>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<keq>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<klt>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kgt>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kle>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kge>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<kne>(const float *A, const float *B, float *out, const float scalar, int size);
template __global__ void kElementWise<ksquared_diff>(const float *A, const float *B, float *out, const float scalar, int size);
template<int operation> __global__ void kElementWise(const float *A, const float *B, float *out, const float scalar, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       switch(operation)
	   {
       	   case kabs: out[i] = fabsf(A[i]); break;
       	   case klog: out[i] = __logf(A[i]); break;
       	   case ksqrt: out[i] = sqrtf(A[i]); break;
       	   case kpow: out[i] = powf(A[i],scalar); break;
       	   case kadd: out[i] = A[i] + B[i]; break;
       	   case ksub: out[i] = A[i] - B[i]; break;
       	   case kdiv: out[i] = fdividef(A[i], B[i]); break;
       	   case kmul: out[i] = A[i] * B[i]; break;
       	   case klogistic: out[i] = 1.0f/(1.0f + expf(A[i])); break;
       	   case klogistic_grad: out[i] = A[i]*(A[i]-1.0f); break;
       	   case krectified: out[i] = A[i] > 0.0f ? A[i] : 0.0f; break;
       	   case krectified_grad: out[i] = A[i] > 0.0f ? 1.0f : 0.0f; break;
       	   case keq: out[i] = (float)(A[i] == B[i]); break;
       	   case klt: out[i] = (float)(A[i] < B[i]); break;
       	   case kgt: out[i] = (float)(A[i] > B[i]); break;
       	   case kge: out[i] = (float)(A[i] >= B[i]); break;
       	   case kle: out[i] = (float)(A[i] <= B[i]); break;
       	   case kne: out[i] = (float)(A[i] != B[i]); break;
       	   case ksquared_diff: out[i] = powf(A[i]-B[i],2.0f); break;
	   }
  }
}

template __global__ void kVectorWise<kvadd>(float *A, float *v, float *out, const float scalar, int rows, int size);
template <int operation> __global__ void kVectorWise(float *A, float *v, float *out, const float scalar, int rows, int size)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	int offset = 0;
	for (unsigned int i = idx;i < size; i += numThreads)
	{
		offset = (i / rows);
		switch(operation)
		{
			case kvadd: out[i] =  A[i] + v[offset]; break;
		}
	}
}


//for column major data
__global__ void kSlice(float *A, float *out, int rows_A, int cols_A, int rstart, int rend, int cstart, int cend)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int rows_out = (rend - rstart);
  int cols_out = (cend - cstart);
  int size = rows_out*cols_out;

  int current_col = 0;
  int offset = 0;
  int current_row = 0;
  for (unsigned int i = idx;i < size; i += numThreads)
  {
	  current_col = i / rows_out;
	  current_row = i - (current_col*rows_out);

	  offset = (rows_A*(current_col+cstart)) + current_row + rstart;
	  out[i] = A[offset];
  }
}



